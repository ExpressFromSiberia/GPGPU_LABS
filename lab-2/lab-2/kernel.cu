#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <time.h>
#include <windows.h>
#include <math.h>
#include <vector>
#include <iostream>
#include <algorithm>
#include <random>

using namespace std;

const unsigned BLOCK_SIZE = 32;
const unsigned int DIM = 2000;

enum MemType
{
    COMMON,
    SHARED,
};

void Matrix_Creator(double* matrix_A, double* matrix_B) {

    std::random_device device;
    std::mt19937 generator(device());
    std::uniform_real_distribution<double> distribution(-30, 30);

    for (int i = 0; i < DIM; i++) {
        for (int j = 0; j < DIM; j++) {
            int	k = DIM * i + j;
            matrix_A[k] = distribution(generator);
            matrix_B[k] = distribution(generator);
        }
    }
}

double Matrix_Subtraction(double* matrix_CP, double* matrix_GP) {

    double max_deviation = 0;

    for (int i = 0; i < DIM * DIM; i++) {
        double deviation = abs(matrix_CP[i] - matrix_GP[i]);
        if (deviation > max_deviation)
            max_deviation = deviation;
    }

    return max_deviation;
}

void Matrix_Multiply_CP(double* Matrix_A, double* Matrix_B, double* Matrix_C) {

    LARGE_INTEGER start, stop, freq;
    QueryPerformanceFrequency(&freq);
    QueryPerformanceCounter(&start);

    for (size_t i = 0; i < DIM; ++i) {
        for (size_t j = 0; j < DIM; ++j) {
            Matrix_C[i * DIM + j] = 0;
            for (size_t k = 0; k < DIM; ++k) {
                Matrix_C[i * DIM + j] += Matrix_A[i * DIM + k] * Matrix_B[k * DIM + j];
            }
        }
    }

    QueryPerformanceCounter(&stop);

    size_t time_delta = stop.QuadPart - start.QuadPart;
    cout << "CPU time: " << static_cast<float>(time_delta) / freq.QuadPart << " seconds" << endl;
}

__global__ void Matrix_Multiply_GP(double* Matrix_A, double* Matrix_B, unsigned int n, double* Matrix_C) {

    double sum = 0.0;
    int   ia = n * blockDim.y * blockIdx.y + n * threadIdx.y;
    int   jb = blockDim.x * blockIdx.x + threadIdx.x;

    if (ia >= n * n || jb >= n)
        return;

    for (int k = 0; k < n; k++) {
        sum += Matrix_A[ia + k] * Matrix_B[jb + k * n];
    }

    Matrix_C[ia + jb] = sum;
}

__global__ void Matrix_Multiply_GP_Shared(double* Matrix_A, double* Matrix_B, unsigned int n, double* Matrix_C)
{
    int subMatrixA_ind = n * BLOCK_SIZE * blockIdx.y;
    int aEnd = subMatrixA_ind + n - 1;

    int aStep = BLOCK_SIZE;
    int subMatrixB_ind = BLOCK_SIZE * blockIdx.x;
    int bStep = BLOCK_SIZE * n;
    double sum = 0.0;

    for (int ia = subMatrixA_ind, ib = subMatrixB_ind; ia <= aEnd; ia += aStep, ib += bStep)
    {
        __shared__ double as[BLOCK_SIZE][BLOCK_SIZE];
        __shared__ double bs[BLOCK_SIZE][BLOCK_SIZE];

        if (subMatrixA_ind + n * threadIdx.y < n * n && ia - subMatrixA_ind + threadIdx.x < n)
            as[threadIdx.y][threadIdx.x] = Matrix_A[ia + n * threadIdx.y + threadIdx.x];
        else
            as[threadIdx.y][threadIdx.x] = 0;
        if (ib - subMatrixB_ind + n * threadIdx.y < n * n && subMatrixB_ind + threadIdx.x < n)
            bs[threadIdx.y][threadIdx.x] = Matrix_B[ib + n * threadIdx.y + threadIdx.x];
        else
            bs[threadIdx.y][threadIdx.x] = 0;

        __syncthreads();

        for (int k = 0; k < BLOCK_SIZE; k++)
            sum += as[threadIdx.y][k] * bs[k][threadIdx.x];

        __syncthreads();
    }

    int ic = n * BLOCK_SIZE * blockIdx.y + BLOCK_SIZE * blockIdx.x;
    if (BLOCK_SIZE * blockIdx.y + threadIdx.y < n && BLOCK_SIZE * blockIdx.x + threadIdx.x < n)
        Matrix_C[ic + n * threadIdx.y + threadIdx.x] = sum;
}

void Matrix_GP_Ligature(double* Matrix_A, double* Matrix_B, double* Matrix_C, MemType type) {

    int numBytes = DIM * DIM * sizeof(double);
    double* adev = NULL;
    double* bdev = NULL;
    double* cdev = NULL;

    hipMalloc((void**)&adev, numBytes);
    hipMalloc((void**)&bdev, numBytes);
    hipMalloc((void**)&cdev, numBytes);

    dim3 threads(BLOCK_SIZE, BLOCK_SIZE);
    dim3 blocks((unsigned int)ceil((double)DIM / threads.x), (unsigned int)ceil((double)DIM / threads.y));
    hipEvent_t start, stop;
    float gpuTime = 0.0f;

    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    hipMemcpy(adev, Matrix_A, numBytes, hipMemcpyHostToDevice);
    hipMemcpy(bdev, Matrix_B, numBytes, hipMemcpyHostToDevice);

    if (type == COMMON)
        Matrix_Multiply_GP << <blocks, threads >> > (adev, bdev, DIM, cdev);
    if (type == SHARED)
        Matrix_Multiply_GP_Shared << <blocks, threads >> > (adev, bdev, DIM, cdev);

    hipMemcpy(Matrix_C, cdev, numBytes, hipMemcpyDeviceToHost);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&gpuTime, start, stop);

    if (type == COMMON)
        cout << "GPU time: " << gpuTime / 1000.0f << " seconds" << endl;
    if (type == SHARED)
        cout << "GPU shared memory time: " << gpuTime / 1000.0f << " seconds" << endl;

    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipFree(adev);
    hipFree(bdev);
    hipFree(cdev);
}

int main(int argc, char* argv[]) {

    double* Matrix_A = new double[DIM * DIM];
    double* Matrix_B = new double[DIM * DIM];
    Matrix_Creator(Matrix_A, Matrix_B);
    double* Matrix_C_CP = new double[DIM * DIM];
    double* Matrix_C_GP = new double[DIM * DIM];
    double* Matrix_C_GP_Shared = new double[DIM * DIM];

    Matrix_Multiply_CP(Matrix_A, Matrix_B, Matrix_C_CP);

    Matrix_GP_Ligature(Matrix_A, Matrix_B, Matrix_C_GP, COMMON);
    double deviation = Matrix_Subtraction(Matrix_C_CP, Matrix_C_GP);
    cout << "Max deviation in matrices values = " << deviation << endl;

    Matrix_GP_Ligature(Matrix_A, Matrix_B, Matrix_C_GP_Shared, SHARED);
    double deviation_shared = Matrix_Subtraction(Matrix_C_CP, Matrix_C_GP_Shared);
    cout << "Max deviation in matrices values = " << deviation_shared << endl;

    delete[] Matrix_A;
    delete[] Matrix_B;
    delete[] Matrix_C_CP;
    delete[] Matrix_C_GP;
    delete[] Matrix_C_GP_Shared;

    return 0;
}